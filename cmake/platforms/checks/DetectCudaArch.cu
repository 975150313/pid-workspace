
#include <hip/hip_runtime.h>
#include <stdio.h>
int main()
{
    int count = 0;
    if (hipSuccess != hipGetDeviceCount(&count)){return -1;}
    if (count == 0) {return -1;}

    printf("%d",count);
    for (int device = 0; device < count; ++device)
    {
        hipDeviceProp_t prop;
        if (hipSuccess != hipGetDeviceProperties(&prop, device)){ continue;}
        printf(";%d.%d", prop.major, prop.minor);
    }
    int driver_version = 0, runtime_version = 0;
    if (hipSuccess != hipDriverGetVersion(&driver_version)){return -1;}
    if (hipSuccess != hipRuntimeGetVersion(&runtime_version)){return -1;}

    printf(";%d;%d", driver_version, runtime_version);
    return 0;
}
